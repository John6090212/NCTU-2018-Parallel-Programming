/**********************************************************************
 * DESCRIPTION:
 *   Serial Concurrent Wave Equation - C Version
 *   This program implements the concurrent wave equation
 *********************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define MAXPOINTS 1000000
#define MAXSTEPS 1000000
#define MINPOINTS 20
#define PI 3.14159265

void check_param(void);
void init_line(void);
__global__ void update (void);
void printfinal (void);

int nsteps,                 	/* number of time steps */
    tpoints, 	     		/* total points along string */
    rcode;                  	/* generic return code */
float  values[MAXPOINTS+2], 	/* values at time t */
       oldval[MAXPOINTS+2], 	/* values at time (t-dt) */
       newval[MAXPOINTS+2]; 	/* values at time (t+dt) */


/**********************************************************************
 *	Checks input values from parameters
 *********************************************************************/
void check_param(void)
{
   char tchar[20];

   /* check number of points, number of iterations */
   while ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS)) {
      printf("Enter number of points along vibrating string [%d-%d]: "
           ,MINPOINTS, MAXPOINTS);
      scanf("%s", tchar);
      tpoints = atoi(tchar);
      if ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS))
         printf("Invalid. Please enter value between %d and %d\n", 
                 MINPOINTS, MAXPOINTS);
   }
   while ((nsteps < 1) || (nsteps > MAXSTEPS)) {
      printf("Enter number of time steps [1-%d]: ", MAXSTEPS);
      scanf("%s", tchar);
      nsteps = atoi(tchar);
      if ((nsteps < 1) || (nsteps > MAXSTEPS))
         printf("Invalid. Please enter value between 1 and %d\n", MAXSTEPS);
   }

   printf("Using points = %d, steps = %d\n", tpoints, nsteps);

}

/**********************************************************************
 *     Initialize points on line
 *********************************************************************/
void init_line(void)
{
   int i, j;
   float x, fac, k, tmp;

   /* Calculate initial values based on sine curve */
   fac = 2.0 * PI;
   k = 0.0; 
   tmp = tpoints - 1;
   for (j = 1; j <= tpoints; j++) {
      x = k/tmp;
      values[j] = sin (fac * x);
      k = k + 1.0;
   } 

   /* Initialize old values array */
   for (i = 1; i <= tpoints; i++) 
      oldval[i] = values[i];
}

/**********************************************************************
 *      Calculate new values using wave equation
 *********************************************************************/
 /*
void do_math(int i)
{
   float dtime, c, dx, tau, sqtau;

   dtime = 0.3;
   c = 1.0;
   dx = 1.0;
   tau = (c * dtime / dx);
   sqtau = tau * tau;
   newval[i] = (2.0 * values[i]) - oldval[i] + (sqtau *  (-2.0)*values[i]);
}
*/

/**********************************************************************
 *     Update all values along line a specified number of times
 *********************************************************************/
__global__ void update(float *GPUoldval,float *GPUvalues,float *GPUnewval,int GPUtpoints,int GPUnsteps)
{
	
   int i;
    int j = blockIdx.x * blockDim.x + threadIdx.x; 
	__shared__ float tempnewval[512];
	__shared__ float tempvalues[512];
	__shared__ float tempoldval[512];

	if(j >= 1 && j <= GPUtpoints){
		tempvalues[threadIdx.x] = GPUvalues[j];
		tempoldval[threadIdx.x] = GPUoldval[j];
	}

	
	
   /* Update values for each time step */
   for (i = 1; i<= GPUnsteps; i++) {
	  
      /* Update points along line for this time step */
      /* global endpoints */
	  if(j >= 1 && j <= GPUtpoints){
		  if ((j == 1) || (j == GPUtpoints))
			tempnewval[threadIdx.x] = 0.0;
		  else{
				float dtime, c, dx, tau, sqtau;
				dtime = 0.3;
				c = 1.0;
				dx = 1.0;
				tau = (c * dtime / dx);
				sqtau = tau * tau;
				tempnewval[threadIdx.x] = (2.0 * tempvalues[threadIdx.x]) - tempoldval[threadIdx.x] + (sqtau *  (-2.0)*tempvalues[threadIdx.x]);
			
		
	      }
	  }

         
      
      /* Update old values with new values */
	  if(j >= 1 && j <= GPUtpoints){
		  tempoldval[threadIdx.x] = tempvalues[threadIdx.x];
          tempvalues[threadIdx.x] = tempnewval[threadIdx.x];
	  }
  
      
   }
   if(j <= GPUtpoints && j >= 1){
	   GPUvalues[j] = tempvalues[threadIdx.x];
   }

   /*
   if(j <= GPUtpoints && j >= 1){
	   printf("threadIdx.x = %d,tempvalues = %f\n",threadIdx.x,tempvalues[threadIdx.x]);
   }
	*/
}

/**********************************************************************
 *     Print final results
 *********************************************************************/
void printfinal()
{
   int i;

   for (i = 1; i <= tpoints; i++) {
      printf("%6.4f ", values[i]);
      if (i%10 == 0)
         printf("\n");
   }
}

/**********************************************************************
 *	Main program
 *********************************************************************/
int main(int argc, char *argv[])
{
	sscanf(argv[1],"%d",&tpoints);
	sscanf(argv[2],"%d",&nsteps);
	check_param();
	printf("Initializing points on the line...\n");
	init_line();
	printf("Updating all points for all time steps...\n");
	float *GPUoldval,*GPUvalues,*GPUnewval;
	int size = MAXPOINTS+2;
	
	hipMalloc(&GPUoldval,size);
	hipMemcpy(GPUoldval,oldval,size,hipMemcpyHostToDevice);
	hipMalloc(&GPUvalues,size);
	hipMemcpy(GPUvalues,values,size,hipMemcpyHostToDevice);
	hipMalloc(&GPUnewval,size);
	hipMemcpy(GPUnewval,newval,size,hipMemcpyHostToDevice);
	//cudaMalloc(&GPUtpoints,1);
	//cudaMemcpy(GPUtpoints,&tpoints,1,cudaMemcpyHostToDevice);
	//cudaMalloc(&GPUnsteps,1);
	//cudaMemcpy(GPUnsteps,&nsteps,1,cudaMemcpyHostToDevice);
	dim3 dimBlock(512);
	dim3 dimGrid(ceil((float)tpoints/512));
	update<<<dimGrid,dimBlock>>>(GPUoldval,GPUvalues,GPUnewval,tpoints,nsteps);
	//update();
	hipMemcpy(values,GPUvalues,size,hipMemcpyDeviceToHost);

	hipFree(GPUoldval);
	hipFree(GPUvalues);
	hipFree(GPUnewval);
	printf("Printing final results...\n");
	printfinal();
	printf("\nDone.\n\n");
	
	return 0;
}
