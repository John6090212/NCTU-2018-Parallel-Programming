/**********************************************************************
 * DESCRIPTION:
 *   Serial Concurrent Wave Equation - C Version
 *   This program implements the concurrent wave equation
 *********************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define MAXPOINTS 1000000
#define MAXSTEPS 1000000
#define MINPOINTS 20
#define PI 3.14159265

void check_param(void);
void init_line(void);
__global__ void update (void);
void printfinal (void);

int nsteps,                 	/* number of time steps */
    tpoints, 	     		/* total points along string */
    rcode;                  	/* generic return code */
float  values[MAXPOINTS+2], 	/* values at time t */
       oldval[MAXPOINTS+2], 	/* values at time (t-dt) */
       newval[MAXPOINTS+2]; 	/* values at time (t+dt) */


/**********************************************************************
 *	Checks input values from parameters
 *********************************************************************/
void check_param(void)
{
   char tchar[20];

   /* check number of points, number of iterations */
   while ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS)) {
      printf("Enter number of points along vibrating string [%d-%d]: "
           ,MINPOINTS, MAXPOINTS);
      scanf("%s", tchar);
      tpoints = atoi(tchar);
      if ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS))
         printf("Invalid. Please enter value between %d and %d\n", 
                 MINPOINTS, MAXPOINTS);
   }
   while ((nsteps < 1) || (nsteps > MAXSTEPS)) {
      printf("Enter number of time steps [1-%d]: ", MAXSTEPS);
      scanf("%s", tchar);
      nsteps = atoi(tchar);
      if ((nsteps < 1) || (nsteps > MAXSTEPS))
         printf("Invalid. Please enter value between 1 and %d\n", MAXSTEPS);
   }

   printf("Using points = %d, steps = %d\n", tpoints, nsteps);

}

/**********************************************************************
 *     Initialize points on line
 *********************************************************************/
void init_line(void)
{
	
   int j;
   float x, fac, k, tmp;

   /* Calculate initial values based on sine curve */
   fac = 2.0 * PI;
   k = 0.0; 
   tmp = tpoints - 1;
   for (j = 1; j <= tpoints; j++) {
      x = k/tmp;
      values[j] = sin (fac * x);
      k = k + 1.0;
   } 

  
}

/**********************************************************************
 *      Calculate new values using wave equation
 *********************************************************************/
 /*
void do_math(int i)
{
   float dtime, c, dx, tau, sqtau;

   dtime = 0.3;
   c = 1.0;
   dx = 1.0;
   tau = (c * dtime / dx);
   sqtau = tau * tau;
   newval[i] = (2.0 * values[i]) - oldval[i] + (sqtau *  (-2.0)*values[i]);
}
*/

/**********************************************************************
 *     Update all values along line a specified number of times
 *********************************************************************/
__global__ void update(float *GPUvalues,int GPUtpoints,int GPUnsteps)
{
	int i;
   int j = blockIdx.x * blockDim.x + threadIdx.x;
   __shared__ float tempnewval[512];
	__shared__ float tempvalues[512];
	__shared__ float tempoldval[512]; 
	if(j >= 1 && j <= GPUtpoints){
		
		
		
		
	
	
		tempvalues[threadIdx.x] = GPUvalues[j];
		tempoldval[threadIdx.x] = tempvalues[threadIdx.x];
	
		
		float dtime, c, dx, tau, sqtau;
		dtime = 0.3;
		c = 1.0;
		dx = 1.0;
		tau = (c * dtime / dx);
		sqtau = tau * tau;
		
		if ((j == 1) || (j == GPUtpoints))
			tempvalues[threadIdx.x] = 0.0;
		
   /* Update values for each time step */
		else{
			for (i = 1; i<= GPUnsteps; i++) {
	  
      /* Update points along line for this time step */
      /* global endpoints */
				
		    
		    
				
				//float compute out of for scope error is very small (-0.80) running time 45s
				//float compute every step error is very small (-0.80) running time is 55s
				tempnewval[threadIdx.x] = ((2.0 - sqtau*2) * tempvalues[threadIdx.x]) - tempoldval[threadIdx.x];
				//error value is bigger running time is 45s
				//tempnewval[threadIdx.x] = 1.82 * tempvalues[threadIdx.x] - tempoldval[threadIdx.x];      
				/* Update old values with new values */
				tempoldval[threadIdx.x] = tempvalues[threadIdx.x];
				tempvalues[threadIdx.x] = tempnewval[threadIdx.x];
			} 

		}
		
	    
    }
	GPUvalues[j] = tempvalues[threadIdx.x];
	/*
	if(j <= GPUtpoints && j >= 1){
			printf("threadIdx.x = %d,tempvalues = %f\n",threadIdx.x,tempvalues[threadIdx.x]);
		}
    */
	
}

/**********************************************************************
 *     Print final results
 *********************************************************************/
void printfinal()
{
   int i;

   for (i = 1; i <= tpoints; i++) {
      printf("%6.4f ", values[i]);
      if (i%10 == 0)
         printf("\n");
   }
}

/**********************************************************************
 *	Main program
 *********************************************************************/
int main(int argc, char *argv[])
{
	sscanf(argv[1],"%d",&tpoints);
	sscanf(argv[2],"%d",&nsteps);
	check_param();
	printf("Initializing points on the line...\n");
	init_line();
	printf("Updating all points for all time steps...\n");
	float *GPUvalues;
	int size = (MAXPOINTS+2) * sizeof(float);
	

	hipMalloc(&GPUvalues,size);
	hipMemcpy(GPUvalues,values,size,hipMemcpyHostToDevice);
	//cudaMalloc(&GPUtpoints,1);
	//cudaMemcpy(GPUtpoints,&tpoints,1,cudaMemcpyHostToDevice);
	//cudaMalloc(&GPUnsteps,1);
	//cudaMemcpy(GPUnsteps,&nsteps,1,cudaMemcpyHostToDevice);
	dim3 dimBlock(512);
	dim3 dimGrid(ceil((float)tpoints/512));
	update<<<dimGrid,dimBlock>>>(GPUvalues,tpoints,nsteps);
	//update();
	hipMemcpy(values,GPUvalues,size,hipMemcpyDeviceToHost);


	hipFree(GPUvalues);

	printf("Printing final results...\n");
	printfinal();
	printf("\nDone.\n\n");
	
	return 0;
}
